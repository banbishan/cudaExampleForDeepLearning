#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void add(int* a, int* b, int* c, int num){
    int i = threadIdx,x;
    if(i < num){
        c[i] = a[i] + b[i];
    }
}

int main(void){
    int num = 10;
    int a[num], b[num], c[num];
    int *a_gpu, *b_gpu, *c_gpu;

    for(int i = 0; i < num; ++i){
        a[i] = i;
        b[i] = i * i;
    }

    hipMalloc((void**)&a_gpu, num * sizeof(int));
    hipMalloc((void**)&b_gpu, num * sizeof(int));
    hipMalloc((void**)&c_gpu, num * sizeof(int));

    hipMemcpy(a_gpu, a, num * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_gpu, b, num * sizeof(int), hipMemcpyHostToDevice);

    add<<<1, num>>>(a_gpu, b_gpu, c_gpu, num);

    hipMemcpy(c, c_gpu, num * sizeof(int), hipMemcpyDeviceToHost);

    for(int i = 0; i < num; ++i){
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    return 0;
}
